#include "hip/hip_runtime.h"
﻿

#include "kernel.h"
#include "hip/hip_runtime_api.h"
#include "memory.h"
#include "iter.h"
#include "parameters.h"

#include <stdio.h>

constexpr int gpuBlockSize = 256;


bool initGPU() {
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    return cudaStatus == hipSuccess;
}


__host__ void getDeviceMemory(void** ptrOut, size_t nBytes) {
    hipError_t cudaStatus = hipMalloc(ptrOut, nBytes);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMalloc failed", cudaStatus);
    }
}

__host__ void freeDeviceMemory(void* ptrIn) {
    hipError_t cudaStatus = hipFree(ptrIn);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipFree failed", cudaStatus);
    }
}

__host__ void copyToGPU(void* devPtr, const void *hostPtr, size_t nBytes) {
    hipError_t cudaStatus = hipMemcpy(devPtr, hostPtr, nBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMemcpy failed", cudaStatus);
    }
}

__host__ void copyToCPU(void* hostPtr, const void* devPtr, size_t nBytes) {
    hipError_t cudaStatus = hipMemcpy(hostPtr, devPtr, nBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMemcpy failed", cudaStatus);
    }
}

void checkAfterKernel() {
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        throw CUDAException("addKernel launch failed", cudaStatus);
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        throw CUDAException("cudaSynchronize", cudaStatus);
    }
}

__constant__ int cudaMaxIter;
__constant__ double cudaBailout;

void setMaxIter(int maxIter) {
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(cudaMaxIter), &maxIter, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CUDAException("setMaxIter", err);
    }
}

void setBailout(double bailout) {
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(cudaBailout), &bailout, sizeof(double), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CUDAException("setBailout", err);
    }
}

__device__ int iterate_z2(const Quat& z0, const Quat& c, int maxIter, double bailout, Quat* orbit) {
    Quat z = z0;
    int iter = 0;
    orbit[0] = z;
    while (iter < maxIter && z.magnitudeSquared() < bailout) {
        z = z.squared() - c;
        orbit[++iter] = z;
    } 
    return iter;
}

__device__ int iterate_z2_no_orbit(const Quat& z0, const Quat& c, int maxIter, double bailout) {
    Quat z = z0;
    int iter = 0;
    while (iter < maxIter && z.magnitudeSquared() < bailout) {
        z = z.squared() - c;
        iter++;
    }
    return iter;
}

__global__ void iterate_0_search_kernel(int N, const Quat* xStartIn, const Quat* cIn, int* resultOut) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N || resultOut[i] == -1) {
        return;
    }
    Quat c = *cIn;
    Quat z = xStartIn[i];
    int iter = 0;
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() - c;
        iter++;
    }
    resultOut[i] = iter;
}

void run_many_Z2_driver(iter_struct& is, const std::vector<Quat>& positions, std::vector<int>& results) {
    setMaxIter(is.maxiter);
    setBailout(is.bailout);
    const int nElems = static_cast<int>(positions.size());
    CUDAStorage<int> resultBuf(nElems);
    CUDAStorage<Quat> c(1);
    c.copyToGPU(&is.c);
    CUDAStorage<Quat> xStart(positions);
    int gridSize = (nElems + gpuBlockSize - 1) / gpuBlockSize;
    iterate_0_search_kernel <<<gridSize, gpuBlockSize>>> (nElems, xStart.devicePtr(), c.devicePtr(), resultBuf.devicePtr()); 
    checkAfterKernel();
    resultBuf.copyToCPU(results);
}

__global__ void object_distance_kernel(int N, const Quat* c, const Quat* xStart, const Quat* zBase, const int(*zLimits)[2], int* zResults) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N || zLimits[i][0] == -1) {
        return;
    }
    for (int z = zLimits[i][0]; z < zLimits[i][1]; z++) {
        Quat start = xStart[i] + z * zBase[i];
        int iter = iterate_z2_no_orbit(start, *c, cudaMaxIter, cudaBailout);
        if (iter == cudaMaxIter) {
            zResults[i] = z;
            return;
        }
    }
    zResults[i] = -1;
}

void row_of_obj_distance_driver(calc_struct& cs, const Quat* positions, const Quat* zBase, const int(*zvals)[2], int* zResults) {
    setMaxIter(cs.f._maxiter);
    setBailout(cs.f._bailout);
    CUDAStorage<Quat> c(1, &cs.f._c);
    CUDAStorage<Quat> xStart(cs.v._xres, positions);
    CUDAStorage<int[2]> zLimits(cs.v._xres, zvals);
    CUDAStorage<Quat> zBaseGPU(cs.v._xres, zBase);
    CUDAStorage<int> zFound(cs.v._xres);
    int gridSize = (cs.v._xres + gpuBlockSize - 1) / gpuBlockSize;
    object_distance_kernel << <gridSize, gpuBlockSize >> > (cs.v._xres, c.devicePtr(), xStart.devicePtr(), zBaseGPU.devicePtr(), zLimits.devicePtr(), zFound.devicePtr());
    checkAfterKernel();
    zFound.copyToCPU(zResults);
}
