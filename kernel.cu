#include "hip/hip_runtime.h"
﻿

#include "kernel.h"

#include <stdio.h>


__host__ void getDeviceMemory(void** ptrOut, size_t nBytes) {
    hipError_t cudaStatus = hipMalloc(ptrOut, nBytes);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMalloc failed", cudaStatus);
    }
}

__host__ void freeDeviceMemory(void* ptrIn) {
    hipError_t cudaStatus = hipFree(ptrIn);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipFree failed", cudaStatus);
    }
}

__host__ void copyToGPU(void* devPtr, void *hostPtr, size_t nBytes) {
    hipError_t cudaStatus = hipMemcpy(devPtr, hostPtr, nBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMemcpy failed", cudaStatus);
    }
}

__host__ void copyToCPU(void* hostPtr, void* devPtr, size_t nBytes) {
    hipError_t cudaStatus = hipMemcpy(hostPtr, devPtr, nBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMemcpy failed", cudaStatus);
    }
}




__global__ void iterate_0_kernel(const Quat* cIn, int* maxIterIn, double* bailoutIn, Quat* __restrict__ orbit, int* resultOut) {
    Quat z = 0;
    orbit[0] = z;
    double zMag2 = z.magnitudeSquared();
    int iter = 0;
    int maxIter = *maxIterIn;
    double bailout = *bailoutIn;
    Quat c = *cIn;
    while (zMag2 < bailout && iter < maxIter) {
        z = z.squared() - c;
        zMag2 = z.magnitudeSquared();
        orbit[++iter] = z;
    }
    *resultOut = iter;
}

int iterate_0_driver(iter_struct* is) {
    CUDAStorage<int> resultBuf(1);
    CUDAStorage<Quat> orbit(is->maxiter);
    CUDAStorage<Quat> c(1);
    CUDAStorage<int> maxiter(1);
    CUDAStorage<double> bailout(1);
    c.copyToGPU(&is->c, 1);
    maxiter.copyToGPU(&is->maxiter, 1);
    bailout.copyToGPU(&is->bailout, 1);
    iterate_0_kernel <<<1, 1>>> (c.ptr(), maxiter.ptr(), bailout.ptr(), orbit.ptr(), resultBuf.ptr());
    int result;
    resultBuf.copyToCPU(&result, 1);
    return result;
}

__global__ void addKernel(int* a, int* b, int* c) {}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
    
    // Allocate GPU buffers for three vectors (two input, one output)
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


bool initGPU() {
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    return cudaStatus == hipSuccess;
}