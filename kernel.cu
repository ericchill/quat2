#include "hip/hip_runtime.h"
﻿

#include "kernel.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "memory.h"
#include "iter.h"
#include "parameters.h"

#include <stdio.h>

#include <crtdbg.h>


bool haveGPU = false;

constexpr int gpuBlockSize = 256;


__host__ bool initGPU(int argc, char** argv) {
    int devID;
    hipDeviceProp_t props;
    hipError_t err;
    haveGPU = false;
    devID = findCudaDevice(argc, (const char**)argv);
    err = hipGetDevice(&devID);
    if (hipSuccess != err) {
        std::cerr << "Couldn't get CUDA device." << std::endl;
        return false;
    }
    err = hipGetDeviceProperties(&props, devID);
    if (hipSuccess != err) {
        std::cerr << "Couldn't get CUDA device properties." << std::endl;
        return false;
    }
    haveGPU = true;
    return true;
}


__host__ void getDeviceMemory(void** ptrOut, size_t nBytes) {
    hipError_t cudaStatus = hipMalloc(ptrOut, nBytes);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMalloc failed", cudaStatus);
    }
}

__host__ void freeDeviceMemory(void* ptrIn) {
    hipError_t cudaStatus = hipFree(ptrIn);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipFree failed", cudaStatus);
    }
}

__host__ void copyToGPU(void* devPtr, const void *hostPtr, size_t nBytes, hipStream_t stream) {
    hipError_t cudaStatus;
    if (0 == stream) {
        cudaStatus = hipMemcpy(devPtr, hostPtr, nBytes, hipMemcpyHostToDevice);
    } else {
        cudaStatus = hipMemcpyAsync(devPtr, hostPtr, nBytes, hipMemcpyHostToDevice, stream);
    }
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMemcpy failed", cudaStatus);
    }
}

__host__ void copyToCPU(void* hostPtr, const void* devPtr, size_t nBytes, hipStream_t stream) {
    hipError_t cudaStatus;
    if (0 == stream) {
        cudaStatus = hipMemcpy(hostPtr, devPtr, nBytes, hipMemcpyDeviceToHost);
    } else {
        cudaStatus = hipMemcpyAsync(hostPtr, devPtr, nBytes, hipMemcpyDeviceToHost, stream);
    }
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipMemcpy failed", cudaStatus);
    }
}

void checkAfterKernel() {
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        throw CUDAException("Kernel launch failed", cudaStatus);
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        throw CUDAException("cudaSynchronize", cudaStatus);
    }
}

__constant__ int cudaMaxIter;
__constant__ int cudaMaxOrbit;
__constant__ double cudaBailout;

void setMaxIter(int maxIter) {
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(cudaMaxIter), &maxIter, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CUDAException("setMaxIter", err);
    }
}

void setMaxOrbit(int maxOrbit) {
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(cudaMaxOrbit), &maxOrbit, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CUDAException("setMaxOrbit", err);
    }
}

void setBailout(double bailout) {
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(cudaBailout), &bailout, sizeof(double), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw CUDAException("setBailout", err);
    }
}


__device__ int iterate_0_cuda(const Quat& z0, const Quat& c, const Quat* p, Quat* orbit) {
    Quat z = z0;
    int iter = 0;
    orbit[0] = z;
    while (iter < cudaMaxOrbit && iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() - c;
        orbit[++iter] = z;
    }
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() - c;
        ++iter;
    }
    orbit[cudaMaxOrbit - 1] = z;
    return iter;
}

__device__ int iterate_0_no_orbit_cuda(const Quat& z0, const Quat& c, const Quat* p) {
    Quat z = z0;
    int iter = 0;
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() - c;
        iter++;
    }
    return iter;
}


__device__ int iterate_1_cuda(const Quat& z0, const Quat& c, const Quat* p, Quat* orbit) {
    Quat z = z0;
    int iter = 0;
    orbit[0] = z;
    while (iter < cudaMaxOrbit && iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = c * z * (1.0 - z);
        orbit[++iter] = z;
    }
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = c * z * (1.0 - z);
        ++iter;
    }
    orbit[cudaMaxOrbit - 1] = z;
    return iter;
}

__device__ int iterate_1_no_orbit_cuda(const Quat& z0, const Quat& c, const Quat* p) {
    Quat z = z0;
    int iter = 0;
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = c * z * (1.0 - z);
        iter++;
    }
    return iter;
}


__device__ int iterate_2_cuda(const Quat& z0, const Quat& c, const Quat* p, Quat* orbit) {
    Quat z = z0;
    int iter = 0;
    orbit[0] = z;
    while (iter < cudaMaxOrbit && iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z * log(z) - c;
        orbit[++iter] = z;
    }
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z * log(z) - c;
        ++iter;
    }
    orbit[cudaMaxOrbit - 1] = z;
    return iter;
}

__device__ int iterate_2_no_orbit_cuda(const Quat& z0, const Quat& c, const Quat* p) {
    Quat z = z0;
    int iter = 0;
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z * log(z) - c;
        iter++;
    }
    return iter;
}


__device__ int iterate_3_cuda(const Quat& z0, const Quat& c, const Quat* p, Quat* orbit) {
    Quat z = z0;
    int iter = 0;
    orbit[0] = z;
    while (iter < cudaMaxOrbit && iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() * z - c;
        orbit[++iter] = z;
    }
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() * z - c;
        ++iter;
    }
    orbit[cudaMaxOrbit - 1] = z;
    return iter;
}

__device__ int iterate_3_no_orbit_cuda(const Quat& z0, const Quat& c, const Quat* p) {
    Quat z = z0;
    int iter = 0;
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() * z - c;
        iter++;
    }
    return iter;
}


__device__ int iterate_4_cuda(const Quat& z0, const Quat& c, const Quat* p, Quat* orbit) {
    Quat z = z0;
    int iter = 0;
    orbit[0] = z;
    while (iter < cudaMaxOrbit && iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() * z - c;
        orbit[++iter] = z;
    }
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() * z - c;
        ++iter;
    }
    orbit[cudaMaxOrbit - 1] = z;
    return iter;
}

__device__ int iterate_4_no_orbit_cuda(const Quat& z0, const Quat& c, const Quat* p) {
    Quat z = z0;
    int iter = 0;
    while (iter < cudaMaxIter && z.magnitudeSquared() < cudaBailout) {
        z = z.squared() * z - c;
        iter++;
    }
    return iter;
}

typedef int (*iterate_fn)(const Quat& z0, const Quat& c, const Quat* p, Quat* orbit);

__device__ iterate_fn iterate_cuda[] = {
    iterate_0_cuda,
    iterate_1_cuda,
    iterate_2_cuda,
    iterate_3_cuda,
    iterate_4_cuda
};

typedef int (*iterate_no_orbit_fn)(const Quat& z0, const Quat& c, const Quat* p);

__device__ iterate_no_orbit_fn iterate_no_orbit_cuda[] = {
    iterate_0_no_orbit_cuda,
    iterate_1_no_orbit_cuda,
    iterate_2_no_orbit_cuda,
    iterate_3_no_orbit_cuda,
    iterate_4_no_orbit_cuda
};


__device__ bool cutaway(const Vec3& x, size_t nCuts, const Vec3* cutNormals, const Vec3* cutPoints) {
    for (unsigned i = 0; i < nCuts; i++) {
        Vec3 y = x - cutPoints[i];
        if (cutNormals[i].dot(y) > 0) {
            return true;
        }
    }
    return false;
}

__global__ void obj_distances_kernel_2(
    size_t N, int formula, int xres, int zres, int antialiasing, const obj_distance_kernel_args* args, const Quat* xStarts,
    Quat* orbits, double* distances, double* lastIters) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    double refinement = 20.0;
    Quat* orbit = &orbits[i * (cudaMaxOrbit + 2)];

    Quat xStart = xStarts[i];
    int iter = -1;
    int z;
    double z2;
    for (z = 0; z < zres && iter != cudaMaxIter; z++) {
        Quat z0 = xStart + static_cast<double>(z) * args->zBase;
        if (!cutaway(Vec3(z0), args->nCuts, args->cutNormals, args->cutPoints)) {
            iter = iterate_no_orbit_cuda[formula](z0, args->c, args->p);
        } else {
            iter = 0;
        }
    }
    double zDouble = static_cast<double>(z);
    if (z < zres) {
        zDouble -= 1.0;
        for (z2 = 1.0; z2 <= refinement && iter == cudaMaxIter; z2 += 1.0) {
            Quat z0 = xStart + (zDouble - z2 / refinement) * args->zBase;
            iter = iterate_cuda[formula](z0, args->c, args->p, orbit);
        }
        z2 -= 2;
    } else {
        z2 = 0;
    }
    distances[i] = floor((zDouble - z2 / refinement) * 1000.0 + 0.5) / 1000.0;
    lastIters[i] = iter;
}

GPURowCalculator::GPURowCalculator(const calc_struct& cs, size_t lBufSize) : _arraySize(lBufSize) {
    hipError_t cudaStatus = hipStreamCreateWithFlags(&_stream, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipStreamCreate failed", cudaStatus);
    }
    cudaStatus = hipStreamCreateWithFlags(&_stream2, hipStreamNonBlocking);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("hipStreamCreate 2 failed", cudaStatus);
    }
    setMaxIter(cs._f._maxiter);
    setBailout(cs._f._bailout);
    setMaxOrbit(cs._f._maxOrbit);
    _kernelArgs.zBase = cs._sbase._z;
    _kernelArgs.c = cs._f._c;
    copyArray(_kernelArgs.p, cs._f._p, 4);
    _kernelArgs.nCuts = cs._cuts.count();
    copyArray(_kernelArgs.cutNormals, cs._cuts.normals(), cs._cuts.maxCuts);
    copyArray(_kernelArgs.cutPoints, cs._cuts.points(), cs._cuts.maxCuts);
    _kernelArgsGPU.copyToGPU(&_kernelArgs, _stream);

    _xStarts = new CUDAStorage<Quat>(lBufSize);
    _orbits = new CUDAStorage<Quat>(lBufSize * (cs._f._maxOrbit + 2));
    _distances = new CUDAStorage<double>(lBufSize);
    _lastIters = new CUDAStorage<double>(lBufSize);
}

GPURowCalculator::~GPURowCalculator() {
    delete _xStarts;
    delete _orbits;
    delete _distances;
    delete _lastIters;
    hipStreamDestroy(_stream);
    hipStreamDestroy(_stream2);
}

void GPURowCalculator::obj_distances(
    calc_struct& cs, size_t N, const Quat* xStarts,
    Quat* orbits, double* distances, double* lastIters) {

    _xStarts->copyToGPU(xStarts, _stream);
    unsigned int gridSize = (static_cast<unsigned int>(N) + gpuBlockSize - 1) / gpuBlockSize;
    obj_distances_kernel_2 << <gridSize, gpuBlockSize, 0, _stream >> > (
        N, cs._f._formula, cs._v._xres, cs._v._zres, cs._v._antialiasing, _kernelArgsGPU.devicePtr(), _xStarts->devicePtr(),
        _orbits->devicePtr(), _distances->devicePtr(), _lastIters->devicePtr());
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        throw CUDAException("obj_distances launch failed", cudaStatus);
    }
    _orbits->copyToCPU(orbits, _stream);
    _distances->copyToCPU(distances, _stream2);
    _lastIters->copyToCPU(lastIters, _stream2);
    cudaStatus = hipStreamSynchronize(_stream);
    cudaStatus = hipStreamSynchronize(_stream2);
    if (cudaStatus != hipSuccess) {
        throw CUDAException("obj_distance stream synchronize failed.", cudaStatus);
    }
}
